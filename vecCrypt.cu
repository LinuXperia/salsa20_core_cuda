/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This software contains source code provided by NVIDIA Corporation.
 * 
 * GPU accelerated Salsa20 Vector crypto core function.
 *
 * This sample demonstrates an implementation of the core Salsa20 crypto function
 * in CTR mode accelerated using CUDA.
 */

// Includes
#include <inttypes.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>

// includes CUDA
#include <hip/hip_runtime.h>

#if defined(__HIPCC__) // NVCC
   #define MY_ALIGN(n) __align__(n)
#elif defined(__GNUC__) // GCC
  #define MY_ALIGN(n) __attribute__((aligned(n)))
#elif defined(_MSC_VER) // MSVC
  #define MY_ALIGN(n) __declspec(align(n))
#else
  #error "Please provide a definition for MY_ALIGN macro for your host compiler!"
#endif

#define ROUNDS 20
#ifndef UINT64_MAX
#define	UINT64_MAX (18446744073709551615ULL)
#endif

#define THREADS_PER_BLOCK (128)
#define XSALSA20_CRYPTO_KEYBYTES 32
#define XSALSA20_CRYPTO_NONCEBYTES 24
#define XSALSA20_BLOCKSZ 64
#define CTR_INBLOCK_SZ (16)
#define CTR_KS_SZ (XSALSA20_BLOCKSZ)
#define BLOCKS_PER_CHUNK_1X 4
#define BLOCKS_PER_CHUNK_2X 1

extern "C" int crypto_stream_salsa20_amd64_xmm6_xor(unsigned char *c, unsigned char *m,
		unsigned long long mlen, unsigned char *n, unsigned char *k);

__constant__ unsigned char MY_ALIGN(sizeof (uint32_t)) key[XSALSA20_CRYPTO_KEYBYTES * THREADS_PER_BLOCK];
__constant__ unsigned char MY_ALIGN(sizeof (uint32_t)) sigma[16];
const unsigned char hsigma[17] = "expand 32-byte k";
unsigned char h_nonce[XSALSA20_CRYPTO_NONCEBYTES];
int pinned = 0;

__host__ __device__ static inline uint32_t
rotate(uint32_t u,int c)
{
  return (u << c) | (u >> (32 - c));
}

__host__ __device__ static inline uint32_t
load_littleendian(const unsigned char *x)
{
  return
      (uint32_t) (x[0]) \
  | (((uint32_t) (x[1])) << 8) \
  | (((uint32_t) (x[2])) << 16) \
  | (((uint32_t) (x[3])) << 24)
  ;
}

__host__ __device__ static inline void
store_littleendian(unsigned char *x, uint32_t u)
{
  x[0] = u; u >>= 8;
  x[1] = u; u >>= 8;
  x[2] = u; u >>= 8;
  x[3] = u;
}

__host__ static inline uint32_t
load_littleendian64(const unsigned char *x)
{
  return
      (uint64_t) (x[0]) \
  | (((uint64_t) (x[1])) << 8) \
  | (((uint64_t) (x[2])) << 16) \
  | (((uint64_t) (x[3])) << 24) \
  | (((uint64_t) (x[4])) << 32) \
  | (((uint64_t) (x[5])) << 40) \
  | (((uint64_t) (x[6])) << 48) \
  | (((uint64_t) (x[7])) << 56)
  ;
}


__host__ static int
crypto_core(
        unsigned char *out,
  const unsigned char *in,
  const unsigned char *k,
  const unsigned char *c
)
{
  uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
  uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
  int i;

  j0 = x0 = load_littleendian(c + 0);
  j1 = x1 = load_littleendian(k + 0);
  j2 = x2 = load_littleendian(k + 4);
  j3 = x3 = load_littleendian(k + 8);
  j4 = x4 = load_littleendian(k + 12);
  j5 = x5 = load_littleendian(c + 4);
  j6 = x6 = load_littleendian(in + 0);
  j7 = x7 = load_littleendian(in + 4);
  j8 = x8 = load_littleendian(in + 8);
  j9 = x9 = load_littleendian(in + 12);
  j10 = x10 = load_littleendian(c + 8);
  j11 = x11 = load_littleendian(k + 16);
  j12 = x12 = load_littleendian(k + 20);
  j13 = x13 = load_littleendian(k + 24);
  j14 = x14 = load_littleendian(k + 28);
  j15 = x15 = load_littleendian(c + 12);

  for (i = ROUNDS;i > 0;i -= 2) {
     x4 ^= rotate( x0+x12, 7);
     x8 ^= rotate( x4+ x0, 9);
    x12 ^= rotate( x8+ x4,13);
     x0 ^= rotate(x12+ x8,18);
     x9 ^= rotate( x5+ x1, 7);
    x13 ^= rotate( x9+ x5, 9);
     x1 ^= rotate(x13+ x9,13);
     x5 ^= rotate( x1+x13,18);
    x14 ^= rotate(x10+ x6, 7);
     x2 ^= rotate(x14+x10, 9);
     x6 ^= rotate( x2+x14,13);
    x10 ^= rotate( x6+ x2,18);
     x3 ^= rotate(x15+x11, 7);
     x7 ^= rotate( x3+x15, 9);
    x11 ^= rotate( x7+ x3,13);
    x15 ^= rotate(x11+ x7,18);
     x1 ^= rotate( x0+ x3, 7);
     x2 ^= rotate( x1+ x0, 9);
     x3 ^= rotate( x2+ x1,13);
     x0 ^= rotate( x3+ x2,18);
     x6 ^= rotate( x5+ x4, 7);
     x7 ^= rotate( x6+ x5, 9);
     x4 ^= rotate( x7+ x6,13);
     x5 ^= rotate( x4+ x7,18);
    x11 ^= rotate(x10+ x9, 7);
     x8 ^= rotate(x11+x10, 9);
     x9 ^= rotate( x8+x11,13);
    x10 ^= rotate( x9+ x8,18);
    x12 ^= rotate(x15+x14, 7);
    x13 ^= rotate(x12+x15, 9);
    x14 ^= rotate(x13+x12,13);
    x15 ^= rotate(x14+x13,18);
  }

  x0 += j0;
  x1 += j1;
  x2 += j2;
  x3 += j3;
  x4 += j4;
  x5 += j5;
  x6 += j6;
  x7 += j7;
  x8 += j8;
  x9 += j9;
  x10 += j10;
  x11 += j11;
  x12 += j12;
  x13 += j13;
  x14 += j14;
  x15 += j15;

  store_littleendian(out + 0,x0);
  store_littleendian(out + 4,x1);
  store_littleendian(out + 8,x2);
  store_littleendian(out + 12,x3);
  store_littleendian(out + 16,x4);
  store_littleendian(out + 20,x5);
  store_littleendian(out + 24,x6);
  store_littleendian(out + 28,x7);
  store_littleendian(out + 32,x8);
  store_littleendian(out + 36,x9);
  store_littleendian(out + 40,x10);
  store_littleendian(out + 44,x11);
  store_littleendian(out + 48,x12);
  store_littleendian(out + 52,x13);
  store_littleendian(out + 56,x14);
  store_littleendian(out + 60,x15);

  return 0;
}

// Variables
unsigned char* h_A = NULL;
unsigned char* h_B = NULL;
unsigned char* d_A = NULL;
bool noprompt = false;

// Functions
void CleanupResources(void);
void Init(unsigned char*, size_t);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	CleanupResources();
        exit(-1);        
    }
}

// This will output the proper error string when calling cudaGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	CleanupResources();
        exit(-1);
    }
}

// end of CUDA Helper Functions


// Device code
__global__ void VecCrypt(unsigned char* A, unsigned int N, uint64_t nblocks, uint64_t p_nonce, int blks_per_chunk)
{
    uint64_t i = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    if (i < N) {
        int k, tot;
        uint32_t *mem;
        uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
        uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
        uint64_t blockno;

        blockno = i*blks_per_chunk;
        tot = (nblocks - blockno > blks_per_chunk) ? blks_per_chunk:(nblocks - blockno);

        for (k = 0; k < tot; k++) {
            j0 = x0 = load_littleendian(sigma + 0);
            j1 = x1 = load_littleendian(key + 0);
            j2 = x2 = load_littleendian(key + 4);
            j3 = x3 = load_littleendian(key + 8);
            j4 = x4 = load_littleendian(key + 12);
            j5 = x5 = load_littleendian(sigma + 4);

            j6 = x6 = p_nonce;
            j7 = x7 = p_nonce >> 32;
            j8 = x8 = blockno;
            j9 = x9 = blockno >> 32;

            j10 = x10 = load_littleendian(sigma + 8);
            j11 = x11 = load_littleendian(key + 16);
            j12 = x12 = load_littleendian(key + 20);
            j13 = x13 = load_littleendian(key + 24);
            j14 = x14 = load_littleendian(key + 28);
            j15 = x15 = load_littleendian(sigma + 12);

            for (i = ROUNDS;i > 0;i -= 2) {
                x4  ^= rotate( x0+x12, 7);
                x8  ^= rotate( x4+ x0, 9);
                x12 ^= rotate( x8+ x4,13);
                x0  ^= rotate(x12+ x8,18);
                x9  ^= rotate( x5+ x1, 7);
                x13 ^= rotate( x9+ x5, 9);
                x1  ^= rotate(x13+ x9,13);
                x5  ^= rotate( x1+x13,18);
                x14 ^= rotate(x10+ x6, 7);
                x2  ^= rotate(x14+x10, 9);
                x6  ^= rotate( x2+x14,13);
                x10 ^= rotate( x6+ x2,18);
                x3  ^= rotate(x15+x11, 7);
                x7  ^= rotate( x3+x15, 9);
                x11 ^= rotate( x7+ x3,13);
                x15 ^= rotate(x11+ x7,18);
                x1  ^= rotate( x0+ x3, 7);
                x2  ^= rotate( x1+ x0, 9);
                x3  ^= rotate( x2+ x1,13);
                x0  ^= rotate( x3+ x2,18);
                x6  ^= rotate( x5+ x4, 7);
                x7  ^= rotate( x6+ x5, 9);
                x4  ^= rotate( x7+ x6,13);
                x5  ^= rotate( x4+ x7,18);
                x11 ^= rotate(x10+ x9, 7);
                x8  ^= rotate(x11+x10, 9);
                x9  ^= rotate( x8+x11,13);
                x10 ^= rotate( x9+ x8,18);
                x12 ^= rotate(x15+x14, 7);
                x13 ^= rotate(x12+x15, 9);
                x14 ^= rotate(x13+x12,13);
                x15 ^= rotate(x14+x13,18);
            }

            x0 += j0;
            x1 += j1;
            x2 += j2;
            x3 += j3;
            x4 += j4;
            x5 += j5;
            x6 += j6;
            x7 += j7;
            x8 += j8;
            x9 += j9;
            x10 += j10;
            x11 += j11;
            x12 += j12;
            x13 += j13;
            x14 += j14;
            x15 += j15;

            mem = (unsigned int *)&A[blockno*XSALSA20_BLOCKSZ];
            *mem ^= x0;  mem++;
            *mem ^= x1;  mem++;
            *mem ^= x2;  mem++;
            *mem ^= x3;  mem++;
            *mem ^= x4;  mem++;
            *mem ^= x5;  mem++;
            *mem ^= x6;  mem++;
            *mem ^= x7;  mem++;
            *mem ^= x8;  mem++;
            *mem ^= x9;  mem++;
            *mem ^= x10;  mem++;
            *mem ^= x11;  mem++;
            *mem ^= x12;  mem++;
            *mem ^= x13;  mem++;
            *mem ^= x14;  mem++;
            *mem ^= x15;
            blockno++;
        }
    }
}

__host__ int
crypto_stream_salsa20_ref_xor(
  unsigned char *m,unsigned long long mlen,
  unsigned char *n,
  unsigned char *k
)
{
  unsigned char in[16];
  unsigned char block[64];
  int i;
  unsigned int u;
  unsigned int blk;

  if (!mlen) return 0;
  blk = 0;

  for (i = 0;i < 8;++i) in[i] = n[i];
  for (i = 8;i < 16;++i) in[i] = 0;

  while (mlen >= XSALSA20_BLOCKSZ) {
    crypto_core(block,in,k,hsigma);
    for (i = 0;i < XSALSA20_BLOCKSZ;++i) m[i] ^= block[i];

    u = 1;
    for (i = 8;i < 16;++i) {
      u += (unsigned int) in[i];
      in[i] = u;
      u >>= 8;
    }

    mlen -= XSALSA20_BLOCKSZ;
    m += XSALSA20_BLOCKSZ;
    blk++;
  }

  if (mlen) {
    crypto_core(block,in,k,hsigma);
    for (i = 0;i < mlen;++i) m[i] ^= block[i];
  }
  return 0;
}

__host__ double
get_wtime_millis(void)
{
    struct timespec ts;
    int rv;

    rv = clock_gettime(CLOCK_MONOTONIC, &ts);
    if (rv == 0)
        return (ts.tv_sec * 1000 + ((double)ts.tv_nsec) / 1000000L);
    return (1);
}

#define	BYTES_TO_MB(x) ((x) / (1024 * 1024))

__host__ double
get_mb_s(uint64_t bytes, double diff)
{
	double bytes_sec;

	bytes_sec = ((double)bytes / diff) * 1000;
	return (BYTES_TO_MB(bytes_sec));
}


// Host code
int main(int argc, char** argv)
{
    printf("Salsa20 Vector Encryption\n");
    unsigned int NBLKS = 4000000, N;
    int rv, blks_per_chunk;
    size_t size, i;
    unsigned char k[32];
    double gpuTime1, gpuTime2, cpuTime1, cpuTime2, strt, en;
    uint64_t v_nonce;
    hipDeviceProp_t deviceProp;

    ParseArguments(argc, argv);
    hipGetDeviceProperties(&deviceProp, 0);
    if (deviceProp.major >= 2)
        blks_per_chunk = BLOCKS_PER_CHUNK_2X;
    else
        blks_per_chunk = BLOCKS_PER_CHUNK_1X;

    N = NBLKS / blks_per_chunk;
    if (NBLKS % blks_per_chunk) N++;
    size = NBLKS * XSALSA20_BLOCKSZ;

    // Allocate input vectors h_A and h_B in host memory
    pinned = 1;
    if (hipHostMalloc(&h_A, size, hipHostMallocDefault) != hipSuccess) {
        pinned = 0;
        h_A = (unsigned char *)malloc(size);
    }
    if (h_A == 0) CleanupResources();
    h_B = (unsigned char *)malloc(size);
    if (h_B == 0) CleanupResources();

    memset(k, 1, XSALSA20_CRYPTO_KEYBYTES);
    memset(h_nonce, 0, XSALSA20_CRYPTO_NONCEBYTES);

    // Initialize input vectors
    printf("Initializing input data\n");
    Init(h_A, size);
    memcpy(h_B, h_A, size);

    // Allocate vectors in device memory
    printf("Allocating device buffer\n");
    checkCudaErrors( hipMalloc((void**)&d_A, size) );

    // Copy vectors from host memory to device memory
    printf("Copying buffer to device\n");

    strt = get_wtime_millis();
    checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(key), k, XSALSA20_CRYPTO_KEYBYTES, 0, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma), hsigma, 16, 0, hipMemcpyHostToDevice) );
    v_nonce = load_littleendian64(h_nonce);
    checkCudaErrors( hipDeviceSynchronize() );

    en = get_wtime_millis();
    gpuTime1 = en - strt;

    printf("Invoking kernel\n");
    strt = get_wtime_millis();

    // Invoke kernel
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecCrypt<<<blocksPerGrid, threadsPerBlock>>>(d_A, N, NBLKS, v_nonce, blks_per_chunk);
    getLastCudaError("kernel launch failure");
    checkCudaErrors( hipDeviceSynchronize() );

    en = get_wtime_millis();
    gpuTime2 = en - strt;

    printf("Copying buffer back to host memory\n");
    // Copy result from device memory to host memory

    strt = get_wtime_millis();
    checkCudaErrors( hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost) );
    checkCudaErrors( hipDeviceSynchronize() );
    en = get_wtime_millis();
    gpuTime1 += (en - strt);
    
    // Verify result
    printf("Computing reference code on CPU\n");
    strt = get_wtime_millis();
    crypto_stream_salsa20_ref_xor(h_B, size, h_nonce + 16, k);
    en = get_wtime_millis();
    cpuTime1 = en - strt;
    rv = 0;

    printf("Verifying result\n");
    for (i = 0; i < size; i++) {
	    if (h_B[i] != h_A[i]) {
		    printf("Byte #%llu differ, %d, %d\n", i, h_B[i], h_A[i]);
		    rv = 1;
		    break;
	    }
    }

    printf("Computing optimized code on CPU\n");
    strt = get_wtime_millis();
    crypto_stream_salsa20_amd64_xmm6_xor(h_B, h_B, size, h_nonce + 16, k);
    en = get_wtime_millis();
    cpuTime2 = en - strt;

    CleanupResources();
    free(h_B);

    if (pinned)
        printf("Data transfer time (pinned mem)         : %f msec\n", gpuTime1);
    else
        printf("Data transfer time (non-pinned mem)     : %f msec\n", gpuTime1);
    printf("GPU computation time                    : %f msec\n", gpuTime2);
    printf("GPU throughput                          : %f MB/s\n", get_mb_s(size, gpuTime2));
    printf("GPU throughput including naive transfer : %f MB/s\n", get_mb_s(size, gpuTime2 + gpuTime1));
    printf("CPU computation time (reference code)   : %f msec\n", cpuTime1);
    printf("CPU throughput (reference code)         : %f MB/s\n", get_mb_s(size, cpuTime1));
    printf("CPU computation time (optimized code)   : %f msec\n", cpuTime2);
    printf("CPU throughput (optimized code)         : %f MB/s\n", get_mb_s(size, cpuTime2));
    if (rv == 0)
        printf("PASSED\n");
    else
        printf("FAILED\n");
}

void CleanupResources(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);

    // Free host memory
    if (h_A) {
        if (pinned)
            hipHostFree(h_A);
        else
            free(h_A);
    }

    hipDeviceReset();
}

// Allocates an array with random float entries.
void Init(unsigned char *data, size_t n)
{
    for (size_t i = 0; i < n; ++i)
        data[i] = i;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--noprompt") == 0 ||
            strcmp(argv[i], "-noprompt") == 0) 
        {
            noprompt = true;
            break;
        }
    }
}
