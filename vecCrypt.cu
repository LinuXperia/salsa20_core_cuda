#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This software contains source code provided by NVIDIA Corporation.
 * 
 * GPU accelerated Salsa20 Vector crypto core function.
 *
 * This sample demonstrates an implementation of the core Salsa20 crypto function
 * in CTR mode accelerated using CUDA.
 */

// Includes
#include <inttypes.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>

// includes, project
#include <sdkHelper.h>  // helper for shared functions common to CUDA SDK samples
#include <shrQATest.h>
#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <cutil_inline.h>

#if defined(__HIPCC__) // NVCC
   #define MY_ALIGN(n) __align__(n)
#elif defined(__GNUC__) // GCC
  #define MY_ALIGN(n) __attribute__((aligned(n)))
#elif defined(_MSC_VER) // MSVC
  #define MY_ALIGN(n) __declspec(align(n))
#else
  #error "Please provide a definition for MY_ALIGN macro for your host compiler!"
#endif

#define ROUNDS 20
#ifndef UINT64_MAX
#define	UINT64_MAX (18446744073709551615ULL)
#endif

#define THREADS_PER_BLOCK (200)
#define XSALSA20_CRYPTO_KEYBYTES 32
#define XSALSA20_CRYPTO_NONCEBYTES 24
#define XSALSA20_BLOCKSZ 64
#define CTR_INBLOCK_SZ (16)
#define CTR_KS_SZ (XSALSA20_BLOCKSZ)
#define BLOCKS_PER_CHUNK 4

extern "C" int crypto_stream_salsa20_amd64_xmm6_xor(unsigned char *c, unsigned char *m,
		unsigned long long mlen, unsigned char *n, unsigned char *k);

__constant__ unsigned char MY_ALIGN(sizeof (uint32_t)) key[XSALSA20_CRYPTO_KEYBYTES * THREADS_PER_BLOCK];
__constant__ unsigned char MY_ALIGN(sizeof (uint32_t)) sigma[16];
const unsigned char hsigma[17] = "expand 32-byte k";
unsigned char h_nonce[XSALSA20_CRYPTO_NONCEBYTES];
int pinned = 0;

__host__ __device__ static inline uint32_t
rotate(uint32_t u,int c)
{
  return (u << c) | (u >> (32 - c));
}

__host__ __device__ static inline uint32_t
load_littleendian(const unsigned char *x)
{
  return
      (uint32_t) (x[0]) \
  | (((uint32_t) (x[1])) << 8) \
  | (((uint32_t) (x[2])) << 16) \
  | (((uint32_t) (x[3])) << 24)
  ;
}

__host__ __device__ static inline void
store_littleendian(unsigned char *x, uint32_t u)
{
  x[0] = u; u >>= 8;
  x[1] = u; u >>= 8;
  x[2] = u; u >>= 8;
  x[3] = u;
}

__host__ static inline uint32_t
load_littleendian64(const unsigned char *x)
{
  return
      (uint64_t) (x[0]) \
  | (((uint64_t) (x[1])) << 8) \
  | (((uint64_t) (x[2])) << 16) \
  | (((uint64_t) (x[3])) << 24) \
  | (((uint64_t) (x[4])) << 32) \
  | (((uint64_t) (x[5])) << 40) \
  | (((uint64_t) (x[6])) << 48) \
  | (((uint64_t) (x[7])) << 56)
  ;
}


__host__ static int
crypto_core(
        unsigned char *out,
  const unsigned char *in,
  const unsigned char *k,
  const unsigned char *c
)
{
  uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
  uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
  int i;

  j0 = x0 = load_littleendian(c + 0);
  j1 = x1 = load_littleendian(k + 0);
  j2 = x2 = load_littleendian(k + 4);
  j3 = x3 = load_littleendian(k + 8);
  j4 = x4 = load_littleendian(k + 12);
  j5 = x5 = load_littleendian(c + 4);
  j6 = x6 = load_littleendian(in + 0);
  j7 = x7 = load_littleendian(in + 4);
  j8 = x8 = load_littleendian(in + 8);
  j9 = x9 = load_littleendian(in + 12);
  j10 = x10 = load_littleendian(c + 8);
  j11 = x11 = load_littleendian(k + 16);
  j12 = x12 = load_littleendian(k + 20);
  j13 = x13 = load_littleendian(k + 24);
  j14 = x14 = load_littleendian(k + 28);
  j15 = x15 = load_littleendian(c + 12);

  for (i = ROUNDS;i > 0;i -= 2) {
     x4 ^= rotate( x0+x12, 7);
     x8 ^= rotate( x4+ x0, 9);
    x12 ^= rotate( x8+ x4,13);
     x0 ^= rotate(x12+ x8,18);
     x9 ^= rotate( x5+ x1, 7);
    x13 ^= rotate( x9+ x5, 9);
     x1 ^= rotate(x13+ x9,13);
     x5 ^= rotate( x1+x13,18);
    x14 ^= rotate(x10+ x6, 7);
     x2 ^= rotate(x14+x10, 9);
     x6 ^= rotate( x2+x14,13);
    x10 ^= rotate( x6+ x2,18);
     x3 ^= rotate(x15+x11, 7);
     x7 ^= rotate( x3+x15, 9);
    x11 ^= rotate( x7+ x3,13);
    x15 ^= rotate(x11+ x7,18);
     x1 ^= rotate( x0+ x3, 7);
     x2 ^= rotate( x1+ x0, 9);
     x3 ^= rotate( x2+ x1,13);
     x0 ^= rotate( x3+ x2,18);
     x6 ^= rotate( x5+ x4, 7);
     x7 ^= rotate( x6+ x5, 9);
     x4 ^= rotate( x7+ x6,13);
     x5 ^= rotate( x4+ x7,18);
    x11 ^= rotate(x10+ x9, 7);
     x8 ^= rotate(x11+x10, 9);
     x9 ^= rotate( x8+x11,13);
    x10 ^= rotate( x9+ x8,18);
    x12 ^= rotate(x15+x14, 7);
    x13 ^= rotate(x12+x15, 9);
    x14 ^= rotate(x13+x12,13);
    x15 ^= rotate(x14+x13,18);
  }

  x0 += j0;
  x1 += j1;
  x2 += j2;
  x3 += j3;
  x4 += j4;
  x5 += j5;
  x6 += j6;
  x7 += j7;
  x8 += j8;
  x9 += j9;
  x10 += j10;
  x11 += j11;
  x12 += j12;
  x13 += j13;
  x14 += j14;
  x15 += j15;

  store_littleendian(out + 0,x0);
  store_littleendian(out + 4,x1);
  store_littleendian(out + 8,x2);
  store_littleendian(out + 12,x3);
  store_littleendian(out + 16,x4);
  store_littleendian(out + 20,x5);
  store_littleendian(out + 24,x6);
  store_littleendian(out + 28,x7);
  store_littleendian(out + 32,x8);
  store_littleendian(out + 36,x9);
  store_littleendian(out + 40,x10);
  store_littleendian(out + 44,x11);
  store_littleendian(out + 48,x12);
  store_littleendian(out + 52,x13);
  store_littleendian(out + 56,x14);
  store_littleendian(out + 60,x15);

  return 0;
}

__device__ static int
crypto_core_device(
        uint32_t *out,
  const uint32_t *in,
  const unsigned char *k,
  const unsigned char *c,
  int stride
)
{
  uint32_t x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;
  uint32_t j0, j1, j2, j3, j4, j5, j6, j7, j8, j9, j10, j11, j12, j13, j14, j15;
  int i;

  j0 = x0 = load_littleendian(c + 0);
  j1 = x1 = load_littleendian(k + 0);
  j2 = x2 = load_littleendian(k + 4);
  j3 = x3 = load_littleendian(k + 8);
  j4 = x4 = load_littleendian(k + 12);
  j5 = x5 = load_littleendian(c + 4);

  j6 = x6 = in[0 * stride];
  j7 = x7 = in[1 * stride];
  j8 = x8 = in[2 * stride];
  j9 = x9 = in[3 * stride];

  j10 = x10 = load_littleendian(c + 8);
  j11 = x11 = load_littleendian(k + 16);
  j12 = x12 = load_littleendian(k + 20);
  j13 = x13 = load_littleendian(k + 24);
  j14 = x14 = load_littleendian(k + 28);
  j15 = x15 = load_littleendian(c + 12);

  for (i = ROUNDS;i > 0;i -= 2) {
     x4 ^= rotate( x0+x12, 7);
     x8 ^= rotate( x4+ x0, 9);
    x12 ^= rotate( x8+ x4,13);
     x0 ^= rotate(x12+ x8,18);
     x9 ^= rotate( x5+ x1, 7);
    x13 ^= rotate( x9+ x5, 9);
     x1 ^= rotate(x13+ x9,13);
     x5 ^= rotate( x1+x13,18);
    x14 ^= rotate(x10+ x6, 7);
     x2 ^= rotate(x14+x10, 9);
     x6 ^= rotate( x2+x14,13);
    x10 ^= rotate( x6+ x2,18);
     x3 ^= rotate(x15+x11, 7);
     x7 ^= rotate( x3+x15, 9);
    x11 ^= rotate( x7+ x3,13);
    x15 ^= rotate(x11+ x7,18);
     x1 ^= rotate( x0+ x3, 7);
     x2 ^= rotate( x1+ x0, 9);
     x3 ^= rotate( x2+ x1,13);
     x0 ^= rotate( x3+ x2,18);
     x6 ^= rotate( x5+ x4, 7);
     x7 ^= rotate( x6+ x5, 9);
     x4 ^= rotate( x7+ x6,13);
     x5 ^= rotate( x4+ x7,18);
    x11 ^= rotate(x10+ x9, 7);
     x8 ^= rotate(x11+x10, 9);
     x9 ^= rotate( x8+x11,13);
    x10 ^= rotate( x9+ x8,18);
    x12 ^= rotate(x15+x14, 7);
    x13 ^= rotate(x12+x15, 9);
    x14 ^= rotate(x13+x12,13);
    x15 ^= rotate(x14+x13,18);
  }

  x0 += j0;
  x1 += j1;
  x2 += j2;
  x3 += j3;
  x4 += j4;
  x5 += j5;
  x6 += j6;
  x7 += j7;
  x8 += j8;
  x9 += j9;
  x10 += j10;
  x11 += j11;
  x12 += j12;
  x13 += j13;
  x14 += j14;
  x15 += j15;

  out[0 * stride] = x0;
  out[1 * stride] = x1;
  out[2 * stride] = x2;
  out[3 * stride] = x3;
  out[4 * stride] = x4;
  out[5 * stride] = x5;
  out[6 * stride] = x6;
  out[7 * stride] = x7;
  out[8 * stride] = x8;
  out[9 * stride] = x9;
  out[10 * stride] = x10;
  out[11 * stride] = x11;
  out[12 * stride] = x12;
  out[13 * stride] = x13;
  out[14 * stride] = x14;
  out[15 * stride] = x15;

  return 0;
}

// Variables
unsigned char* h_A = NULL;
unsigned char* h_B = NULL;
unsigned char* d_A = NULL;
bool noprompt = false;

// Functions
void CleanupResources(void);
void Init(unsigned char*, size_t);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	CleanupResources();
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	CleanupResources();
        exit(-1);
    }
}

// end of CUDA Helper Functions


// Device code
__global__ void VecCrypt(unsigned char* A, unsigned int N, uint64_t nblocks, uint64_t p_nonce)
{
    uint64_t i = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x, blockno;
    __shared__ unsigned char MY_ALIGN(sizeof (uint32_t)) __in[CTR_INBLOCK_SZ  * THREADS_PER_BLOCK];
    __shared__ unsigned char MY_ALIGN(sizeof (uint32_t)) __block[CTR_KS_SZ  * THREADS_PER_BLOCK];
    uint32_t *block, *in;
    uint32_t val1, *val2;

    if (i < N) {
        int k, tot;
        int j;

        in = (uint32_t *)&__in[threadIdx.x * sizeof (uint32_t)];
        block = (uint32_t *)&__block[threadIdx.x * sizeof (uint32_t)];
        in[0 * THREADS_PER_BLOCK] = p_nonce;
        in[1 * THREADS_PER_BLOCK] = (p_nonce >> 32);
        in[2 * THREADS_PER_BLOCK] = 0;
        in[3 * THREADS_PER_BLOCK] = 0;

        blockno = i*BLOCKS_PER_CHUNK;
        tot = (nblocks - blockno > BLOCKS_PER_CHUNK) ? BLOCKS_PER_CHUNK:(nblocks - blockno);

        for (k = 0; k < tot; k++) {
            in[2 * THREADS_PER_BLOCK] = blockno;
            in[3 * THREADS_PER_BLOCK] = (blockno >> 32);

            crypto_core_device(block,in,key,sigma, THREADS_PER_BLOCK);

            for (j = 0;j < XSALSA20_BLOCKSZ; j+= sizeof (uint32_t)) {
                val1 = block[j/(sizeof (uint32_t)) * THREADS_PER_BLOCK];
                val2 = (unsigned int *)&A[blockno*XSALSA20_BLOCKSZ + j];
                *val2 ^= val1;
            }
            blockno++;
        }
    }
}

__host__ int
crypto_stream_salsa20_ref_xor(
  unsigned char *m,unsigned long long mlen,
  unsigned char *n,
  unsigned char *k
)
{
  unsigned char in[16];
  unsigned char block[64];
  int i;
  unsigned int u;
  unsigned int blk;

  if (!mlen) return 0;
  blk = 0;

  for (i = 0;i < 8;++i) in[i] = n[i];
  for (i = 8;i < 16;++i) in[i] = 0;

  while (mlen >= XSALSA20_BLOCKSZ) {
    crypto_core(block,in,k,hsigma);
    for (i = 0;i < XSALSA20_BLOCKSZ;++i) m[i] ^= block[i];

    u = 1;
    for (i = 8;i < 16;++i) {
      u += (unsigned int) in[i];
      in[i] = u;
      u >>= 8;
    }

    mlen -= XSALSA20_BLOCKSZ;
    m += XSALSA20_BLOCKSZ;
    blk++;
  }

  if (mlen) {
    crypto_core(block,in,k,hsigma);
    for (i = 0;i < mlen;++i) m[i] ^= block[i];
  }
  return 0;
}

__host__ double
get_wtime_millis(void)
{
    struct timespec ts;
    int rv;

    rv = clock_gettime(CLOCK_MONOTONIC, &ts);
    if (rv == 0)
        return (ts.tv_sec * 1000 + ((double)ts.tv_nsec) / 1000000L);
    return (1);
}

#define	BYTES_TO_MB(x) ((x) / (1024 * 1024))

__host__ double
get_mb_s(uint64_t bytes, double diff)
{
	double bytes_sec;

	bytes_sec = ((double)bytes / diff) * 1000;
	return (BYTES_TO_MB(bytes_sec));
}


// Host code
int main(int argc, char** argv)
{
    shrQAStart(argc, argv);

    printf("Vector Encryption\n");
    unsigned int NBLKS = 4000000, N;
    int rv;
    size_t size, i;
    unsigned char k[32];
    double gpuTime1, gpuTime2, cpuTime1, cpuTime2, strt, en;
    unsigned int hTimer;
    uint64_t v_nonce;

    ParseArguments(argc, argv);

    N = NBLKS / BLOCKS_PER_CHUNK;
    if (NBLKS % BLOCKS_PER_CHUNK) N++;
    size = NBLKS * XSALSA20_BLOCKSZ;

    // Allocate input vectors h_A and h_B in host memory
    pinned = 1;
    if (hipHostMalloc(&h_A, size) != hipSuccess) {
        pinned = 0;
        h_A = (unsigned char *)malloc(size);
    }
    if (h_A == 0) CleanupResources();
    h_B = (unsigned char *)malloc(size);
    if (h_B == 0) CleanupResources();

    memset(k, 1, XSALSA20_CRYPTO_KEYBYTES);
    memset(h_nonce, 0, XSALSA20_CRYPTO_NONCEBYTES);

    // Initialize input vectors
    printf("Initializing input data\n");
    Init(h_A, size);
    memcpy(h_B, h_A, size);
    cutilCheckError( cutCreateTimer(&hTimer) );

    // Allocate vectors in device memory
    printf("Allocating device buffer\n");
    checkCudaErrors( hipMalloc((void**)&d_A, size) );

    // Copy vectors from host memory to device memory
    printf("Copying buffer to device\n");
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );

    checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(key), k, XSALSA20_CRYPTO_KEYBYTES, 0, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma), hsigma, 16, 0, hipMemcpyHostToDevice) );
    v_nonce = load_littleendian64(h_nonce);
    checkCudaErrors( hipDeviceSynchronize() );

    cutilCheckError( cutStopTimer(hTimer) );
    gpuTime1 = cutGetTimerValue(hTimer);

    printf("Invoking kernel\n");
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );

    // Invoke kernel
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecCrypt<<<blocksPerGrid, threadsPerBlock>>>(d_A, N, NBLKS, v_nonce);
    getLastCudaError("kernel launch failure");
    checkCudaErrors( hipDeviceSynchronize() );

    cutilCheckError( cutStopTimer(hTimer) );
    gpuTime2 = cutGetTimerValue(hTimer);

    printf("Copying buffer back to host memory\n");
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cutilCheckError( cutResetTimer(hTimer) );
    cutilCheckError( cutStartTimer(hTimer) );

    checkCudaErrors( hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost) );
    checkCudaErrors( hipDeviceSynchronize() );

    cutilCheckError( cutStopTimer(hTimer) );
    gpuTime1 += cutGetTimerValue(hTimer);
    
    printf("Verifying result\n");
    // Verify result
    strt = get_wtime_millis();
    crypto_stream_salsa20_ref_xor(h_B, size, h_nonce + 16, k);
    en = get_wtime_millis();
    cpuTime1 = en - strt;
    rv = 0;
    for (i = 0; i < size; i++) {
	    if (h_B[i] != h_A[i]) {
		    printf("Byte #%llu differ, %d, %d\n", i, h_B[i], h_A[i]);
		    rv = 1;
		    break;
	    }
    }

    strt = get_wtime_millis();
    crypto_stream_salsa20_amd64_xmm6_xor(h_B, h_B, size, h_nonce + 16, k);
    en = get_wtime_millis();
    cpuTime2 = en - strt;

    CleanupResources();
    free(h_B);

    if (pinned)
        printf("Data transfer time (pinned mem)         : %f msec\n", gpuTime1);
    else
        printf("Data transfer time (non-pinned mem)     : %f msec\n", gpuTime1);
    printf("GPU computation time                    : %f msec\n", gpuTime2);
    printf("GPU throughput                          : %f MB/s\n", get_mb_s(size, gpuTime2));
    printf("GPU throughput including naive transfer : %f MB/s\n", get_mb_s(size, gpuTime2 + gpuTime1));
    printf("CPU computation time (reference code)   : %f msec\n", cpuTime1);
    printf("CPU throughput (reference code)         : %f MB/s\n", get_mb_s(size, cpuTime1));
    printf("CPU computation time (optimized code)   : %f msec\n", cpuTime2);
    printf("CPU throughput (optimized code)         : %f MB/s\n", get_mb_s(size, cpuTime2));
    shrQAFinishExit(argc, (const char **)argv, (rv==0) ? QA_PASSED : QA_FAILED);
}

void CleanupResources(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);

    // Free host memory
    if (h_A) {
        if (pinned)
            hipHostFree(h_A);
        else
            free(h_A);
    }

    hipDeviceReset();
}

// Allocates an array with random float entries.
void Init(unsigned char *data, size_t n)
{
    for (size_t i = 0; i < n; ++i)
        data[i] = i;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--noprompt") == 0 ||
            strcmp(argv[i], "-noprompt") == 0) 
        {
            noprompt = true;
            break;
        }
    }
}
